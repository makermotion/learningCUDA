#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1000

__global__ void square() { printf("%d\n", threadIdx.x * threadIdx.x); }

int main() {
  square<<<1, N>>>();
  hipDeviceSynchronize();
  return 0;
}
