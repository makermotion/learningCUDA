#include <hip/hip_runtime.h>
#include <stdio.h>
#define msg "Hello, World!\n"

__global__ void hello_from_gpu(void) { printf(msg); }

int main() {
  hello_from_gpu<<<1, 32>>>();
  hipDeviceSynchronize();
  return 0;
}
