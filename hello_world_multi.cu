#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void helloWorld1() { printf("Hello, world 1!\n"); }
__global__ void helloWorld2() { printf("Hello, world 2!\n"); }

int main() {
  helloWorld1<<<1, 1>>>();
  helloWorld2<<<1, 1>>>();
  hipDeviceSynchronize();
  printf("on CPU");
  return 0;
}
