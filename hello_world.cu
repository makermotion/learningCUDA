#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void helloWorld() { printf("Hello, world!\n"); }

int main() {
  helloWorld<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}
