#include <hip/hip_runtime.h>  // Include the CUDA runtime API
#include <stdio.h> // Include the standard I/O library for printf
#define N 100      // Define a constant N with value 100

// Kernel function to square the index of the thread
__global__ void square(int *a) {
  int idx = threadIdx.x;
  a[idx] = idx * idx;
}

int main() {
  int a[N], *dev_a; // Declare an array 'a' of size N and a pointer 'dev_a' for
                    // device memory
  int i;            // Declare an integer 'i' for loop iteration
  hipError_t err;  // Declare a variable to hold CUDA error codes

  // Allocate memory on the device for 'dev_a' of size N integers
  err = hipMalloc(&dev_a, N * sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device memory: %s\n",
            hipGetErrorString(err));
    return 1;
  }

  // Launch the kernel with 1 block and N threads
  square<<<1, N>>>(dev_a);
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch kernel: %s\n", hipGetErrorString(err));
    hipFree(dev_a);
    return 1;
  }

  // Copy the data from device memory to host memory
  err = hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy data from device to host: %s\n",
            hipGetErrorString(err));
    hipFree(dev_a);
    return 1;
  }

  // Loop through the array 'a' and print each element
  for (i = 0; i < N; i++) {
    printf("%d\n", a[i]);
  }

  // Free the allocated device memory
  hipFree(dev_a);

  return 0; // Return 0 to indicate successful completion
}
